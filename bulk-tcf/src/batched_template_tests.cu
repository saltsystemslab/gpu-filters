#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "bulk_tcf.cuh"
#include "bulk_tcf_metadata.cuh"

#include <openssl/rand.h>


#define COUNTING_CYCLES 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void check_hits(bool * hits, uint64_t * misses, uint64_t nitems){


	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= nitems) return;

	if (!hits[tid]){

		atomicAdd((unsigned long long int *) misses, 1ULL);

	}
}

template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> split_insert_timing(bulk_tcf<Key, Val, Wrapper> * my_tcf, uint64_t * large_keys, key_val_pair<Key, Val, Wrapper> * keys, uint64_t nvals, uint64_t * misses){


	uint64_t num_blocks = my_tcf->get_num_blocks();

	uint64_t num_teams = my_tcf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();

	my_tcf->attach_lossy_buffers(large_keys, keys, nvals, num_blocks);


	hipDeviceSynchronize();
	
	gpuErrchk( hipPeekAtLastError() );


	auto midpoint = std::chrono::high_resolution_clock::now();


	my_tcf->bulk_insert(misses, num_teams);
	

	hipDeviceSynchronize();

	gpuErrchk( hipPeekAtLastError() );
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


	std::chrono::duration<double> attach_diff = midpoint-start;
  	std::chrono::duration<double> insert_diff = end-midpoint;	
  	std::chrono::duration<double> diff = end-start;



  	std::cout << "attached in " << attach_diff.count() << ", inserted in " << insert_diff.count() << ".\n";

  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}


template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> split_insert_timing_cycles(bulk_tcf<Key, Val, Wrapper> * my_tcf, uint64_t * large_keys, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses, uint64_t * cycles, uint64_t * num_warps){


	uint64_t num_blocks = my_tcf->get_num_blocks();

	uint64_t num_teams = my_tcf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();

	my_tcf->attach_lossy_buffers_cycles(large_keys, vals, nvals, num_blocks, cycles, num_warps);


	hipDeviceSynchronize();
	
	gpuErrchk( hipPeekAtLastError() );


	auto midpoint = std::chrono::high_resolution_clock::now();


	my_tcf->bulk_insert_cycles(misses, cycles, num_teams, num_warps);
	

	hipDeviceSynchronize();

	gpuErrchk( hipPeekAtLastError() );
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


	std::chrono::duration<double> attach_diff = midpoint-start;
  	std::chrono::duration<double> insert_diff = end-midpoint;	
  	std::chrono::duration<double> diff = end-start;



  	std::cout << "attached in " << attach_diff.count() << ", inserted in " << insert_diff.count() << ".\n";

  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}



template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> bulk_query_timing(bulk_tcf<Key, Val, Wrapper> * my_tcf, uint64_t * large_keys, key_val_pair<Key, Val, Wrapper> * keys, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));


	uint64_t num_blocks = my_tcf->get_num_blocks();

	uint64_t num_teams = my_tcf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	my_tcf->attach_lossy_buffers(large_keys, keys, nvals, num_blocks);
	my_tcf->bulk_query(hits, num_teams);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}


template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> fp_timing(bulk_tcf<Key, Val, Wrapper> * my_tcf, uint64_t * large_keys, key_val_pair<Key, Val, Wrapper> * keys, uint64_t nvals, uint64_t * misses){




	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));


	uint64_t num_blocks = my_tcf->get_num_blocks();

	uint64_t num_teams = my_tcf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	my_tcf->attach_lossy_buffers(large_keys, keys, nvals, num_blocks);
	my_tcf->bulk_query(hits, num_teams);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);



	//check hits

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "FP Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("FP Sorted Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu, ratio: %f\n", misses[0], 1.0 * (nvals - misses[0])/nvals);  

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

   return diff;
}

template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> bulk_delete_timing(bulk_tcf<Key, Val, Wrapper> * my_tcf, uint64_t * large_keys, key_val_pair<Key, Val, Wrapper> * keys, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));


	uint64_t num_blocks = my_tcf->get_num_blocks();

	uint64_t num_teams = my_tcf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	my_tcf->attach_lossy_buffers(large_keys, keys, nvals, num_blocks);
	my_tcf->bulk_delete(hits, num_teams);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Deleted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Deletes per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}


template <typename T>
__host__ T * generate_data(uint64_t nitems){


	//malloc space

	T * vals = (T *) malloc(nitems * sizeof(T));


	//			   100,000,000
	uint64_t cap = 100000000ULL;

	for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

		uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


		RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



		to_fill += togen;

		printf("Generated %llu/%llu\n", to_fill, nitems);

	}

	return vals;
}

template <typename T>
__host__ T * load_main_data(uint64_t nitems){


	char main_location[] = "/global/cscratch1/sd/hunterm/tcf_data/main_data-32-data.txt";

	//char main_location[] = "/pscratch/sd/h/hunterm/tcf_data/main_data-32-data.txt";

	char * vals = (char * ) malloc(nitems * sizeof(T));

	//std::ifstream myfile(main_location);

	//std::string line;


	FILE * pFile;


	pFile = fopen(main_location, "rb");

	if (pFile == NULL) abort();

	size_t result;

	result = fread(vals, 1, nitems*sizeof(T), pFile);

	if (result != nitems*sizeof(T)) abort();



	// //current supported format is no spacing one endl for the file terminator.
	// if (myfile.is_open()){


	// 	getline(myfile, line);

	// 	strncpy(vals, line.c_str(), sizeof(uint64_t)*nitems);

	// 	myfile.close();
		

	// } else {

	// 	abort();
	// }


	return (T *) vals;


}

template <typename T>
__host__ T * load_alt_data(uint64_t nitems){


	char main_location[] = "/global/cscratch1/sd/hunterm/tcf_data/fp_data-32-data.txt";

	//char main_location[] = "/pscratch/sd/h/hunterm/tcf_data/fp_data-32-data.txt";


	char * vals = (char * ) malloc(nitems * sizeof(T));


	//std::ifstream myfile(main_location);

	//std::string line;


	FILE * pFile;


	pFile = fopen(main_location, "rb");

	if (pFile == NULL) abort();

	size_t result;

	result = fread(vals, 1, nitems*sizeof(T), pFile);

	if (result != nitems*sizeof(T)) abort();



	return (T *) vals;


}

int main(int argc, char** argv) {
	

	uint64_t nbits = atoi(argv[1]);

	uint64_t num_batches = atoi(argv[2]);

	double batch_percent = 1.0 / num_batches;


	uint64_t nitems = (1ULL << nbits) * .85;


	//add one? just to guarantee that the clip is correct
	uint64_t items_per_batch = 1.05*nitems * batch_percent;


	printf("Starting test with %d bits, %llu items inserted in %d batches of %d.\n", nbits, nitems, num_batches, items_per_batch);


	//swap this to remainder type
	using key_type = uint16_t;
	using main_data_type = key_val_pair<key_type>;

	uint64_t * keys;
	uint64_t * dev_keys;


	//main_data_type * vals;
	main_data_type * short_keys;


	//comment this out
	keys = generate_data<uint64_t>(nitems);

	//keys = load_main_data<main_data_type>(nitems);


	uint64_t * fp_keys;

	//main_data_type * fp_vals;

	//generate fp data to see comparison with true inserts
	fp_keys = generate_data<uint64_t>(nitems);




	
	hipMalloc((void ** )& short_keys, items_per_batch*sizeof(main_data_type));

	hipMalloc((void ** )& dev_keys, items_per_batch*sizeof(uint64_t));

	//hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);


	//bool * inserts;


	// hipMalloc((void ** )& inserts, items_per_batch*sizeof(bool));

	// hipMemset(inserts, 0, items_per_batch*sizeof(bool));



	// hipMalloc((void ** )& dev_other_vals, nitems*sizeof(other_vals[0]));

	// hipMemcpy(dev_other_vals, other_vals, nitems * sizeof(other_vals[0]), hipMemcpyHostToDevice);


	//allocate misses counter
	uint64_t * misses;
	hipMallocManaged((void **)& misses, sizeof(uint64_t));

	misses[0] = 0;

	#if COUNTING_CYCLES

	uint64_t * cycles;

	hipMallocManaged((void**)&cycles, 8*sizeof(uint64_t));

	cycles[0] = 0;
	cycles[1] = 0;
	cycles[2] = 0;
	cycles[3] = 0;
	cycles[4] = 0;
	cycles[5] = 0;
	cycles[6] = 0;
	cycles[7] = 0;


	uint64_t * num_warps;

	hipMallocManaged((void**)&num_warps, 6*sizeof(double));

	hipDeviceSynchronize();


	#endif


	//change the way tcf is built to better suit test and use cases? TODO with active reconstruction for exact values / struct support
	
	//quad_hash_table * ht =  build_hash_table(1ULL << nbits);
	bulk_tcf<key_type> * tcf = build_tcf<key_type>( (uint64_t)(1ULL << nbits));

	//std::chrono::duration<double> diff = std::chrono::nanoseconds::zero();


	std::chrono::duration<double>  * insert_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	std::chrono::duration<double>  * query_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	std::chrono::duration<double>  * fp_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	std::chrono::duration<double>  * delete_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	
	uint64_t * batch_amount = (uint64_t *) malloc(num_batches*sizeof(uint64_t));





	printf("Setup done\n");

	//wipe_vals<<<nitems/32+1, 32>>>(dev_vals, nitems);


	hipDeviceSynchronize();

	

	for (int batch = 0; batch< num_batches; batch++){

		//calculate size of segment

		printf("Batch %d:\n", batch);

		//runs from batch/num_batches*nitems to batch
		uint64_t start = batch*nitems/num_batches;
		uint64_t end = (batch+1)*nitems/num_batches;
		if (end > nitems) end = nitems;

		uint64_t items_to_insert = end-start;


		assert(items_to_insert <= items_per_batch);

		batch_amount[batch] = items_to_insert;

		//prep dev_vals for this round

		hipMemcpy(dev_keys, keys + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		//hipMemcpy(dev_vals, vals + start, items_to_insert*sizeof(main_data_type), hipMemcpyHostToDevice);

		hipDeviceSynchronize();

		//launch inserts

		#if COUNTING_CYCLES

		if (batch == 0){
			insert_diff[batch] = split_insert_timing_cycles<key_type>(tcf, dev_keys, short_keys, items_to_insert, misses, cycles, num_warps);

		} else {
			insert_diff[batch] = split_insert_timing<key_type>(tcf, dev_keys, short_keys, items_to_insert, misses);

		}
		

		#else

		insert_diff[batch] = split_insert_timing<key_type>(tcf, dev_keys, short_keys, items_to_insert, misses);

		#endif

		
		hipDeviceSynchronize();

		hipMemcpy(dev_keys, keys + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);


		hipDeviceSynchronize();


		//launch queries
		query_diff[batch] = bulk_query_timing<key_type>(tcf, dev_keys, short_keys, items_to_insert, misses);


		hipDeviceSynchronize();

		hipMemcpy(dev_keys, fp_keys + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		//hipMemcpy(dev_vals, fp_vals + start, items_to_insert*sizeof(main_data_type), hipMemcpyHostToDevice);

		hipDeviceSynchronize();


		//false queries
		fp_diff[batch] = fp_timing<key_type>(tcf, dev_keys, short_keys, items_to_insert, misses);


		hipDeviceSynchronize();



		hipMemcpy(dev_keys, keys + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);


		hipDeviceSynchronize();


		//launch queries
		delete_diff[batch] = bulk_delete_timing<key_type>(tcf, dev_keys, short_keys, items_to_insert/2, misses);


		hipDeviceSynchronize();


		hipMemcpy(dev_keys, keys + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		hipDeviceSynchronize();

		query_diff[batch] = bulk_query_timing<key_type>(tcf, dev_keys+items_to_insert/2, short_keys, items_to_insert/2, misses);




		//keep some organized spacing
		printf("\n\n");

		fflush(stdout);

		hipDeviceSynchronize();



	}

	std::chrono::duration<double> summed_insert_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_insert_diff += insert_diff[i];
	}

	std::chrono::duration<double> summed_query_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_query_diff += query_diff[i];
	}

	std::chrono::duration<double> summed_fp_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_fp_diff += fp_diff[i];
	}

	std::chrono::duration<double> summed_delete_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_delete_diff += delete_diff[i];
	}

	printf("Tests Finished.\n");

	std::cout << "Queried " << nitems << " in " << summed_insert_diff.count() << " seconds\n";

	printf("Final speed: %f\n", nitems/summed_insert_diff.count());

	if (argc == 4){

		printf("Dumping into file\n");

		const char * dir = "batched_results/";

		char filename_insert[256];
		char filename_lookup[256];
		char filename_false_lookup[256];
		char filename_delete[256];
		char filename_aggregate[256];

		const char * insert_op = "_insert_";

		snprintf(filename_insert, strlen(dir) + strlen(argv[3]) + strlen(insert_op) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], insert_op, argv[1], argv[2]);

		const char * lookup_op = "_lookup_";

		snprintf(filename_lookup, strlen(dir) + strlen(argv[3]) + strlen(lookup_op) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], lookup_op, argv[1], argv[2]);

		const char * fp_ops = "_fp_";

		snprintf(filename_false_lookup, strlen(dir) + strlen(argv[3]) + strlen(fp_ops) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], fp_ops, argv[1], argv[2]);

		const char * delete_op = "_delete_";

		snprintf(filename_delete, strlen(dir) + strlen(argv[3]) + strlen(fp_ops) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], delete_op, argv[1], argv[2]);


		const char * agg_ops = "_aggregate_";

		snprintf(filename_aggregate, strlen(dir) + strlen(argv[3]) + strlen(agg_ops)+ strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], agg_ops, argv[1], argv[2]);


		FILE *fp_insert = fopen(filename_insert, "w");
		FILE *fp_lookup = fopen(filename_lookup, "w");
		FILE *fp_false_lookup = fopen(filename_false_lookup, "w");
		FILE *fp_delete = fopen(filename_delete, "w");
		FILE *fp_agg = fopen(filename_aggregate, "w");

		if (fp_insert == NULL) {
			printf("Can't open the data file %s\n", filename_insert);
			exit(1);
		}

		if (fp_lookup == NULL ) {
		    printf("Can't open the data file %s\n", filename_lookup);
			exit(1);
		}

		if (fp_false_lookup == NULL) {
			printf("Can't open the data file %s\n", filename_false_lookup);
			exit(1);
		}

		if (fp_delete == NULL) {
			printf("Can't open the data file %s\n", filename_delete);
			exit(1);
		}

		if (fp_agg == NULL) {
			printf("Can't open the data file %s\n", filename_aggregate);
			exit(1);
		}


		printf("Writing results to file: %s\n",  filename_insert);

		fprintf(fp_insert, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_insert, "%d", i*100/num_batches);

			fprintf(fp_insert, " %f\n", batch_amount[i]/insert_diff[i].count());
		}
		printf("Insert performance written!\n");

		fclose(fp_insert);


		printf("Writing results to file: %s\n",  filename_lookup);

		fprintf(fp_lookup, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_lookup, "%d", i*100/num_batches);

			fprintf(fp_lookup, " %f\n", batch_amount[i]/query_diff[i].count());
		}
		printf("lookup performance written!\n");

		fclose(fp_lookup);



		printf("Writing results to file: %s\n",  filename_false_lookup);

		fprintf(fp_false_lookup, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_false_lookup, "%d", i*100/num_batches);

			fprintf(fp_false_lookup, " %f\n", batch_amount[i]/fp_diff[i].count());
		}
		printf("false_lookup performance written!\n");

		fclose(fp_false_lookup);


		printf("Writing results to file: %s\n",  filename_delete);

		fprintf(fp_delete, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_delete, "%d", i*100/num_batches);

			fprintf(fp_delete, " %f\n", batch_amount[i]/fp_diff[i].count());
		}
		printf("Delete performance written!\n");

		fclose(fp_delete);


		printf("Writing results to file: %s\n",  filename_aggregate);

		//fprintf(fp_agg, "x_0 y_0\n");

		fprintf(fp_agg, "Aggregate inserts: %f\n", nitems/summed_insert_diff.count());
		fprintf(fp_agg, "Aggregate Queries: %f\n", nitems/summed_query_diff.count());
		fprintf(fp_agg, "Aggregate fp: %f\n", nitems/summed_fp_diff.count());



		printf("false_lookup performance written!\n");

		fclose(fp_agg);



	}


	#if COUNTING_CYCLES

	printf("Cycle counts: insert_total, load, distribute, sorting, merging, hashing, set_buffer, set_len\n%llu, %llu, %llu, %llu, %llu, %llu, %llu, %llu\n", cycles[0], cycles[6], cycles[7], cycles[1], cycles[2], cycles[3], cycles[4], cycles[5]);



	printf("%f, %f, %f, %f, %f, %f, %f, %f\n", 1.0*cycles[0]/num_warps[0], 1.0*cycles[6]/num_warps[0], 1.0*cycles[7]/num_warps[0], 1.0*cycles[1]/num_warps[0], 1.0*cycles[2]/num_warps[0], 1.0*cycles[3]/num_warps[1], 1.0*cycles[4]/num_warps[2], 1.0*cycles[5]/num_warps[3]);
	


	// printf("Hashing, set buffer, set len, load, distribute, sorting, merging");
	// printf("%f, %f, %f, %f, %f, %f, %f, %f\n",  )
	hipFree(cycles);

	hipFree(num_warps);
	#endif


	free(keys);

	free(fp_keys);

	hipFree(dev_keys);
	hipFree(short_keys);

	hipFree(misses);

	free_tcf(tcf);

	
	//free counters	

	return 0;

}
