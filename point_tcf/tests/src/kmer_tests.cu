#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 *
 *        About:
 *          This file contains k-mer speed tests for several Hash Table Types
 *          built using POGGERS. For more verbose testing please see the 
 *          benchmarks folder.
 *
 * ============================================================================
 */




//#include "include/templated_quad_table.cuh"
#include <poggers/metadata.cuh>
#include <poggers/hash_schemes/murmurhash.cuh>
#include <poggers/probing_schemes/linear_probing.cuh>
#include <poggers/probing_schemes/double_hashing.cuh>
#include <poggers/probing_schemes/power_of_two.cuh>
#include <poggers/insert_schemes/single_slot_insert.cuh>
#include <poggers/insert_schemes/bucket_insert.cuh>
#include <poggers/insert_schemes/power_of_n.cuh>
#include <poggers/representations/key_val_pair.cuh>
#include <poggers/representations/shortened_key_val_pair.cuh>
#include <poggers/sizing/default_sizing.cuh>
#include <poggers/tables/base_table.cuh>

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <openssl/rand.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>



template <size_t size_in_bytes> struct kmer {

 char data [size_in_bytes];


 operator int() const { return data[0]; }

 kmer& operator=(int i){
    data[0] = (char) i;
    return *this;
  }

};


template <std::size_t size_in_bytes>
__host__ __device__ bool operator==(kmer<size_in_bytes> A, kmer<size_in_bytes> B){

   for (int i=0; i < size_in_bytes; i++){

      if (A.data[i] != B.data[i]) return false;
   }

   return true;

}


using key_type = kmer<20>;
using val_type = uint64_t;



//using insert_type = poggers::insert_schemes::single_slot_insert<uint64_t, uint64_t, 8, 8, poggers::representations::key_val_pair, 5, poggers::hashers::murmurHasher, poggers::probing_schemes::doubleHasher>;

using table_type = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::bucket_insert, 200, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
     // poggers::representations::key_val_pair, 8>

     //using forst_tier_table_type = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, poggers::insert_schemes::single_slot_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
    
using second_tier_table_type = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::single_slot_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher, true, table_type>;

using inner_table = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::bucket_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

using small_double_type = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::bucket_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher, true, inner_table>;

//using p2_table = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::shortened_key_val_wrapper<uint16_t>::key_val_pair, 8, 32, poggers::insert_schemes::power_of_n_insert_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   
using tier_one_iceberg = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 64, poggers::insert_schemes::bucket_insert, 1, poggers::probing_schemes::linearProber, poggers::hashers::murmurHasher>;

using tier_two_icerberg = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::power_of_n_insert_scheme, 2, poggers::probing_schemes::powerOfTwoHasher, poggers::hashers::murmurHasher>;

using tier_three_iceberg = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::bucket_insert, 10, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


using tier_two_icerberg_joined = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 8, poggers::insert_schemes::power_of_n_insert_scheme, 2, poggers::probing_schemes::powerOfTwoHasher, poggers::hashers::murmurHasher>;

using iceberg_table = poggers::tables::static_table<key_type, val_type, poggers::representations::key_val_pair, 8, 64, poggers::insert_schemes::bucket_insert, 1, poggers::probing_schemes::linearProber, poggers::hashers::murmurHasher, true, tier_two_icerberg_joined>;



#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <typename T>
__host__ T * generate_data(uint64_t nitems){


   //malloc space

   T * vals = (T *) malloc(nitems * sizeof(T));


   //          100,000,000
   uint64_t cap = 100000000ULL;

   for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

      uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


      RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



      to_fill += togen;

      //printf("Generated %llu/%llu\n", to_fill, nitems);

   }

   return vals;
}


template <typename Filter, typename Key, typename Val>
__global__ void print_tid_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals){


      auto tile = filter->get_my_tile();

      uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

      if (tid >= nvals) return;


      if (tile.thread_rank() == 0) printf("%llu: %d, %d\n", tid, keys[tid], vals[tid]);

}

template <typename Filter, typename Key, typename Val>
__global__ void speed_insert_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;




   if (!filter->insert(tile, keys[tid], vals[tid]) && tile.thread_rank() == 0){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   } 
      //else{

   //    Val test_val = 0;
   //    assert(filter->query(tile, keys[tid], test_val));
   // }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

template <typename Filter, typename Key, typename Val>
__global__ void speed_insert_kernel_one_thread(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid2 = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid2 > 0) return;


   for (uint64_t tid=0; tid < nvals; tid++){

      if (tid % 10 == 0 && tile.thread_rank() == 0) printf("%llu\n", tid);

      if (!filter->insert(tile, keys[tid], vals[tid]) && tile.thread_rank() == 0){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   } else{

      Val test_val = 0;
      assert(filter->query(tile, keys[tid], test_val));
   }


   }





   //assert(filter->insert(tile, keys[tid], vals[tid]));


}



template <typename Filter, typename Key, typename Val>
__global__ void speed_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   Val val = 0;
   //val += 0;

   if (!filter->query(tile,keys[tid], val) && tile.thread_rank() == 0){
      atomicAdd((unsigned long long int *) query_misses, 1ULL);
   } else {

      if (val != vals[tid] && tile.thread_rank() == 0){
         atomicAdd((unsigned long long int *) query_failures, 1ULL);
      }

   }
   //assert(filter->query(tile, keys[tid], val));


}

template <typename Filter, typename Key, typename Val, typename Sizing_Type>
__host__ void test_speed(Sizing_Type * Initializer){

   uint64_t nitems = Initializer->total()*.9;

   Key * host_keys = generate_data<Key>(nitems);
   Val * host_vals = generate_data<Val>(nitems);

   Key * dev_keys;

   Val * dev_vals;

   hipMalloc((void **)& dev_keys, nitems*sizeof(Key));
   hipMalloc((void **)& dev_vals, nitems*sizeof(Val));

   hipMemcpy(dev_keys, host_keys, nitems*sizeof(Key), hipMemcpyHostToDevice);
   hipMemcpy(dev_vals, host_vals, nitems*sizeof(Val), hipMemcpyHostToDevice);


   uint64_t * misses;

   hipMallocManaged((void **)& misses, sizeof(uint64_t)*3);
   hipDeviceSynchronize();

   misses[0] = 0;
   misses[1] = 0;
   misses[2] = 0;

   //static seed for testing
   Filter * test_filter = Filter::generate_on_device(Initializer, 42);

   hipDeviceSynchronize();

   //print_tid_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(nitems),test_filter->get_block_size(nitems)>>>(test_filter, dev_keys, dev_vals, nitems);

   hipDeviceSynchronize();

   auto insert_start = std::chrono::high_resolution_clock::now();

   //add function for configure parameters - should be called by ht and return dim3
   speed_insert_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(nitems),test_filter->get_block_size(nitems)>>>(test_filter, dev_keys, dev_vals, nitems, misses);
   hipDeviceSynchronize();
   auto insert_end = std::chrono::high_resolution_clock::now();


   hipMemcpy(dev_keys, host_keys, nitems*sizeof(Key), hipMemcpyHostToDevice);
   hipMemcpy(dev_vals, host_vals, nitems*sizeof(Val), hipMemcpyHostToDevice);


   hipDeviceSynchronize();

   auto query_start = std::chrono::high_resolution_clock::now();

   speed_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(nitems),test_filter->get_block_size(nitems)>>>(test_filter, dev_keys, dev_vals, nitems, &misses[1], &misses[2]);
   hipDeviceSynchronize();
   auto query_end = std::chrono::high_resolution_clock::now();




   std::chrono::duration<double> insert_diff = insert_end-insert_start;
   std::chrono::duration<double> query_diff = query_end-query_start;


   hipDeviceSynchronize();
   std::cout << "Inserted " << nitems << " in " << insert_diff.count() << " seconds\n";
   std::cout << "Queried " << nitems << " in " << query_diff.count() << " seconds\n";

   printf("Inserts/Queries: %f / %f\n", 1.0*nitems/insert_diff.count(), 1.0*nitems/query_diff.count());
   printf("%llu / %llu / %llu\n", misses[0], misses[1], misses[2]);

   hipDeviceSynchronize();

   hipFree(misses);

   hipDeviceSynchronize();

   hipFree(dev_keys);
   hipFree(dev_vals);

   Filter::free_on_device(test_filter);

   free(host_keys);
   free(host_vals);

}


int main(int argc, char** argv) {

   // poggers::sizing::size_in_num_slots<1> first_size_20(1ULL << 20);
   // printf("2^20\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_20);

   // poggers::sizing::size_in_num_slots<1> first_size_22(1ULL << 22);
   // printf("2^22\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_22);

   // poggers::sizing::size_in_num_slots<1> first_size_24(1ULL << 24);
   // printf("2^24\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_24);

   // poggers::sizing::size_in_num_slots<1> first_size_26(1ULL << 26);
   // printf("2^26\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_26);

   // poggers::sizing::size_in_num_slots<1> first_size_28(1ULL << 28);
   // printf("2^28\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_28);


   // printf("alt table\n");

   // poggers::sizing::size_in_num_slots<1>half_split_20(6000);
   // test_speed<p2_table, key_type, val_type>(&half_split_20);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_22);

   // poggers::sizing::size_in_num_slots<2>half_split_24(1ULL << 23, 1ULL << 23);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_24);

   // poggers::sizing::size_in_num_slots<2>half_split_26(1ULL << 25, 1ULL << 25);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_26);


//   printf("P2 tiny table\n");
   // poggers::sizing::size_in_num_slots<1>half_split_28(1ULL << 28);
   // test_speed<p2_table, key_type, val_type>(&half_split_28);


   printf("Double Hashing\n");

   poggers::sizing::size_in_num_slots<1> double_hashing_size(1ULL << 10);
   test_speed<table_type, key_type, val_type>(&double_hashing_size);

   printf("Icerberg - Tier one\n");


   poggers::sizing::size_in_num_slots<1> tier_one_iceberg_size(1ULL << 10);
   test_speed<tier_one_iceberg, key_type, val_type>(&tier_one_iceberg_size);


   printf("Icerberg - Tier two\n");

   //this section is allocated 1/8th of the space as tier one
   poggers::sizing::size_in_num_slots<1> tier_two_iceberg_size((1ULL << 10)/8);
   test_speed<tier_two_icerberg, key_type, val_type>(&tier_two_iceberg_size);

   printf("Icerberg - Tier three\n");

   poggers::sizing::size_in_num_slots<1> tier_three_iceberg_size((1500));
   test_speed<tier_three_iceberg, key_type, val_type>(&tier_three_iceberg_size);


   printf("Icerberg - Joined\n");

   poggers::sizing::size_in_num_slots<3> iceberg_size((1ULL << 10), (1ULL << 10)/8, 1500);
   test_speed<iceberg_table, key_type, val_type>(&iceberg_size);


	return 0;

}
