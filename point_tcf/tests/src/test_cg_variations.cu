#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 *
 *        About:
 *          This file contains speed tests for several Hash Table Types
 *          built using POGGERS. For more verbose testing please see the 
 *          benchmarks folder.
 *
 * ============================================================================
 */




//#include "include/templated_quad_table.cuh"
#include <poggers/metadata.cuh>
#include <poggers/hash_schemes/murmurhash.cuh>
#include <poggers/probing_schemes/linear_probing.cuh>
#include <poggers/probing_schemes/double_hashing.cuh>
#include <poggers/probing_schemes/power_of_two.cuh>
#include <poggers/insert_schemes/single_slot_insert.cuh>
#include <poggers/insert_schemes/bucket_insert.cuh>
#include <poggers/insert_schemes/power_of_n.cuh>
#include <poggers/representations/key_val_pair.cuh>
#include <poggers/representations/shortened_key_val_pair.cuh>
#include <poggers/sizing/default_sizing.cuh>
#include <poggers/tables/base_table.cuh>
#include <poggers/insert_schemes/power_of_n_shortcut.cuh>

#include <poggers/sizing/variadic_sizing.cuh>

#include <poggers/representations/soa.cuh>
#include <poggers/insert_schemes/power_of_n_shortcut_buckets.cuh>

#include <poggers/tables/bucketed_table.cuh>


#include <poggers/representations/12_bit_bucket.cuh>
#include <poggers/insert_schemes/power_of_n_shortcut_buckets.cuh>
#include <poggers/representations/dynamic_container.cuh>
#include <poggers/representations/key_only.cuh>


#include <poggers/insert_schemes/grouped_power_buckets.cuh>

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <openssl/rand.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


//using tiny_static_table_4 = poggers::tables::static_table<uint64_t, uint16_t, poggers::representations::shortened_key_val_wrapper<uint16_t>::key_val_pair, 4, 4, poggers::insert_schemes::bucket_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
//using tiny_static_table_4 = poggers::tables::bucketed_table<uint64_t, uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 4, 4, poggers::insert_schemes::bucket_insert, 20, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

//simple test version
 using tcf_twelve = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_bucket_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//test version with old buckets
//this works
//using tcf = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::struct_of_arrays, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_bucket_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//12-16
//using tcf = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_bucket_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

//12-8
//using tcf_twelve_8 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 8, poggers::insert_schemes::blocked_bucket_insert<10>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//12-12
//using tcf_twelve_12 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 8, poggers::insert_schemes::blocked_bucket_insert<6>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//12-32
//using tcf_twelve_32 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//using tcf_no_back = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//using tcf_16_16 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

//using tcf_16_32 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 8, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


//using warpcore_bloom = warpcore::BloomFilter<uint64_t>;

//using tcf_8_8 = poggers::tables::static_table<uint64_t,uint8_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint8_t>::representation, 4, 8, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


uint64_t num_slots_per_p2(uint64_t nitems){

   //uint64_t nitems = .9*(1ULL << nbits);

   //for p=1/100, this is the correct value

   uint64_t nslots = 959*nitems/100;
   printf("using %llu slots\n", nslots);
   return nslots; 

}


template <typename T>
__host__ T * load_main_data(uint64_t nitems){


   char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/main_data-32-data.txt";

   //char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/main_data-32-data.txt";

   char * vals = (char * ) malloc(nitems * sizeof(T));

   //std::ifstream myfile(main_location);

   //std::string line;


   FILE * pFile;


   pFile = fopen(main_location, "rb");

   if (pFile == NULL) abort();

   size_t result;

   result = fread(vals, 1, nitems*sizeof(T), pFile);

   if (result != nitems*sizeof(T)) abort();



   // //current supported format is no spacing one endl for the file terminator.
   // if (myfile.is_open()){


   //    getline(myfile, line);

   //    strncpy(vals, line.c_str(), sizeof(uint64_t)*nitems);

   //    myfile.close();
      

   // } else {

   //    abort();
   // }


   return (T *) vals;


}

template <typename T>
__host__ T * load_alt_data(uint64_t nitems){


   char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/fp_data-32-data.txt";

   //char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/fp_data-32-data.txt";


   char * vals = (char * ) malloc(nitems * sizeof(T));


   //std::ifstream myfile(main_location);

   //std::string line;


   FILE * pFile;


   pFile = fopen(main_location, "rb");

   if (pFile == NULL) abort();

   size_t result;

   result = fread(vals, 1, nitems*sizeof(T), pFile);

   if (result != nitems*sizeof(T)) abort();



   return (T *) vals;


}

template <typename T>
__host__ T * generate_data(uint64_t nitems){


   //malloc space

   T * vals = (T *) malloc(nitems * sizeof(T));


   //          100,000,000
   uint64_t cap = 100000000ULL;

   for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

      uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


      RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



      to_fill += togen;

      //printf("Generated %llu/%llu\n", to_fill, nitems);

   }

   return vals;
}


template <typename Filter, typename Key, typename Val>
__global__ void find_first_fill(Filter * filter, Key * keys, Val * vals, uint64_t nitems, uint64_t * returned_nitems){


   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid != 0) return;

   // if (tile.thread_rank() == 0){

   //    for (int i = 0; i < 10; i++){
   //       printf("%d: %llu, %llu\n", i, keys[i], vals[i]);
   //    }
   // }


   //printf("Starting!\n");

   for (uint64_t i = 0; i < nitems; i++){


      if (!filter->insert(tile, keys[i])){

         if (tile.thread_rank() == 0){

            printf("Inserted %llu / %llu, %f full\n", i, nitems, 1.0*i/nitems);

         }

         returned_nitems[0] = i;

         return;

      } else {

         Val alt_val = 0;
         assert(filter->query(tile, keys[i], alt_val));
         assert(alt_val == vals[i]);


      }

      
   }

   if (tile.thread_rank() == 0) printf("All %llu items inserted\n", nitems);

}



template <typename Filter, typename Key, typename Val>
__global__ void speed_insert_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;


   if (!filter->insert(tile, keys[tid], vals[tid])){

   //    if (tile.thread_rank() == 0)
   //    atomicAdd((unsigned long long int *) misses, 1ULL);


   // } else{

   //    Val test_val = 0;
   //    assert(filter->query(tile, keys[tid], test_val));

   //    //assert(test_val == vals[tid]);
   }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

template <typename Filter, typename Key, typename Val>
__global__ void debug_insert_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses, bool * missed){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;


   if (!filter->insert(tile, keys[tid], vals[tid])){

      //filter->insert(tile, keys[tid], vals[tid]);

      if (tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) misses, 1ULL);

      missed[tid] = true;


   } else{

      Val test_val = 0;
      assert(filter->query(tile, keys[tid], test_val));

      missed[tid] = false;

      //assert(test_val == vals[tid]);
   }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

template <typename Filter, typename Key, typename Val>
__global__ void debug_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures, bool * missed){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   if (missed[tid]) return;

   Val test_val = 0;

   if (!filter->query(tile,keys[tid], test_val)){


      filter->query(tile,keys[tid], test_val);


      if(tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) query_misses, 1ULL);

   } else {


      // if (test_val != vals[tid] && tile.thread_rank() == 0){
      //    atomicAdd((unsigned long long int *) query_failures, 1ULL);
      // }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Key, typename Val>
__global__ void speed_remove_kernel(Filter * filter, Key * keys, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;




   if (!filter->remove(tile, keys[tid]) && tile.thread_rank() == 0){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   } 
      //else{

   //    Val test_val = 0;
   //    assert(filter->query(tile, keys[tid], test_val));
   // }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

__global__ void count_bf_misses(bool * vals, uint64_t nitems, uint64_t * misses){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= nitems) return;


   if (!vals[tid]){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   }
}

template <typename Filter, typename Key, typename Val>
__global__ void speed_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   Val test_val = 0;

   if (!filter->query(tile,keys[tid], test_val)){


   //    filter->query(tile,keys[tid], test_val);


   //    if(tile.thread_rank() == 0)
   //    atomicAdd((unsigned long long int *) query_misses, 1ULL);

   // } else {


      // if (test_val != vals[tid] && tile.thread_rank() == 0){
      //    atomicAdd((unsigned long long int *) query_failures, 1ULL);
      // }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Key, typename Val>
__global__ void fp_speed_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   Val test_val = 0;

   if (!filter->query(tile,keys[tid], test_val)){


   //    filter->query(tile,keys[tid], test_val);


      if(tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) query_misses, 1ULL);

   // } else {


      // if (test_val != vals[tid] && tile.thread_rank() == 0){
      //    atomicAdd((unsigned long long int *) query_failures, 1ULL);
      // }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Val>
__host__ void test_tcf_speed(const std::string& filename, int num_bits, int num_batches){


   using Key = uint64_t;
   //using Val = uint8_t;

   //using Filter = tcf;

   //std::cout << "Starting " << filename << " " << num_bits << std::endl;

   // poggers::sizing::size_in_num_slots<2> pre_init ((1ULL << num_bits), (1ULL << num_bits)/100);

   // poggers::sizing::size_in_num_slots<2> * Initializer = &pre_init;


   poggers::sizing::size_in_num_slots<1> pre_init ((1ULL << num_bits));

   poggers::sizing::size_in_num_slots<1> * Initializer = &pre_init;



   uint64_t nitems = Initializer->total()*.9;

   Key * host_keys = generate_data<Key>(nitems);
   Val * host_vals = generate_data<Val>(nitems);


   Key * fp_keys = generate_data<Key>(nitems);

   Key * dev_keys;

   Val * dev_vals;




   uint64_t * misses;

   hipMallocManaged((void **)& misses, sizeof(uint64_t)*5);
   hipDeviceSynchronize();

   //printf("Data generated\n");

   misses[0] = 0;
   misses[1] = 0;
   misses[2] = 0;
   misses[3] = 0;
   misses[4] = 0;

   //static seed for testing
   Filter * test_filter = Filter::generate_on_device(Initializer, 42);

   hipDeviceSynchronize();

   //init timing materials
   std::chrono::duration<double>  * insert_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
   std::chrono::duration<double>  * query_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
   std::chrono::duration<double>  * fp_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));

   std::chrono::duration<double>  * delete_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));



   uint64_t * batch_amount = (uint64_t *) malloc(num_batches*sizeof(uint64_t));

   //print_tid_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(nitems),test_filter->get_block_size(nitems)>>>(test_filter, dev_keys, dev_vals, nitems);


   for (uint64_t i = 0; i < num_batches; i++){

      uint64_t start_of_batch = i*nitems/num_batches;
      uint64_t items_in_this_batch = (i+1)*nitems/num_batches;

      if (items_in_this_batch > nitems) items_in_this_batch = nitems;

      items_in_this_batch = items_in_this_batch - start_of_batch;


      batch_amount[i] = items_in_this_batch;


      hipMalloc((void **)& dev_keys, items_in_this_batch*sizeof(Key));
      hipMalloc((void **)& dev_vals, items_in_this_batch*sizeof(Val));


      hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      bool * missed;

      hipMalloc((void **)&missed, items_in_this_batch*sizeof(bool));



      //ensure GPU is caught up for next task
      hipDeviceSynchronize();

      auto insert_start = std::chrono::high_resolution_clock::now();

      //add function for configure parameters - should be called by ht and return dim3
      speed_insert_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, misses);
      //debug_insert_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, misses, missed);
      
      hipDeviceSynchronize();
      auto insert_end = std::chrono::high_resolution_clock::now();

      insert_diff[i] = insert_end-insert_start;

      hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      hipDeviceSynchronize();

      auto query_start = std::chrono::high_resolution_clock::now();

      speed_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[1], &misses[2]);
      //debug_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[1], &misses[2], missed);
      
      hipDeviceSynchronize();
      auto query_end = std::chrono::high_resolution_clock::now();


     
      query_diff[i] = query_end - query_start;

      hipMemcpy(dev_keys, fp_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      hipDeviceSynchronize();

      auto fp_start = std::chrono::high_resolution_clock::now();

      speed_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[3], &misses[4]);


      hipDeviceSynchronize();
      auto fp_end = std::chrono::high_resolution_clock::now();

      fp_diff[i] = fp_end-fp_start;


      hipFree(dev_keys);
      hipFree(dev_vals);

      hipFree(missed);


   }

   //deletes
   // for (uint64_t i = 0; i < num_batches; i++){

   //    uint64_t start_of_batch = i*nitems/num_batches;
   //    uint64_t items_in_this_batch = (i+1)*nitems/num_batches;

   //    if (items_in_this_batch > nitems) items_in_this_batch = nitems;

   //    items_in_this_batch = items_in_this_batch - start_of_batch;


   //   // batch_amount[i] = items_in_this_batch;


   //    hipMalloc((void **)& dev_keys, items_in_this_batch*sizeof(Key));
   //    //hipMalloc((void **)& dev_vals, items_in_this_batch*sizeof(Val));


   //    hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
   //    //hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);

   //    hipDeviceSynchronize();

   //    auto delete_start = std::chrono::high_resolution_clock::now();

   //    speed_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(items_in_this_batch),test_filter->get_block_size(items_in_this_batch)>>>(test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[1], &misses[2]);
   //    hipDeviceSynchronize();
   //    auto delete_end = std::chrono::high_resolution_clock::now();


     
   //    delete_diff[i] = delete_end - delete_start;

   // }

   hipDeviceSynchronize();


   Filter::free_on_device(test_filter);

   free(host_keys);
   free(host_vals);
   free(fp_keys);

   //free pieces

   //time to output


   //printf("nitems: %llu, insert misses: %llu, query missed: %llu, query wrong %llu, fp missed %llu, fp wrong %llu\n", nitems, misses[0], misses[1], misses[2], misses[3], misses[4]);

   std::chrono::duration<double> summed_insert_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_insert_diff += insert_diff[i];
   }

   std::chrono::duration<double> summed_query_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_query_diff += query_diff[i];
   }

   std::chrono::duration<double> summed_fp_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_fp_diff += fp_diff[i];
   }

   std::chrono::duration<double> summed_delete_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_delete_diff += delete_diff[i];
   }

  


   const uint64_t scaling_factor = 1000000ULL;



   double insert_throughput = nitems/(scaling_factor*summed_insert_diff.count());
      
   double lookup_throughput = nitems/(scaling_factor*summed_query_diff.count());

   double fp_throughput = nitems/(scaling_factor*summed_fp_diff.count());

   std::cout << "Name,  insert perf (M/s), lookup perf (M/s), FP perf (M/s)" << std::endl;
   std::cout << filename << ": " << insert_throughput << ", " << lookup_throughput << ", " << fp_throughput <<"." << std::endl;

   // std::cout << insert_file << std::endl;
   return;




}


template <typename Filter, typename Key, typename Val>
__host__ void tcf_find_first_fill(uint64_t num_bits){


   //std::cout << "Starting " << filename << " " << num_bits << std::endl;

   poggers::sizing::size_in_num_slots<1> pre_init ((1ULL << num_bits));

   poggers::sizing::size_in_num_slots<1> * Initializer = &pre_init;


   // poggers::sizing::size_in_num_slots<2> pre_init ((1ULL << num_bits), (1ULL << num_bits)/100);


   //  poggers::sizing::size_in_num_slots<2> * Initializer = &pre_init;

   // poggers::sizing::size_in_num_slots<1> pre_init ((1ULL << num_bits));

   // poggers::sizing::size_in_num_slots<1> * Initializer = &pre_init;



   uint64_t nitems = Initializer->total();

   Key * host_keys = generate_data<Key>(nitems);
   Val * host_vals = generate_data<Val>(nitems);


   Key * dev_keys;
   Val * dev_vals;


   // printf("Host keys\n");
   // for (int i = 0; i < 10; i++){
   //       printf("%d: %llu, %llu\n", i, host_keys[i], host_vals[i]);
   //    }

   uint64_t * misses;

   hipMallocManaged((void ** )&misses, sizeof(uint64_t)*2);

   misses[0] = 0;
   misses[1] = 0;

   uint64_t * returned_nitems;
   hipMallocManaged((void **)&returned_nitems, sizeof(uint64_t));  

   returned_nitems[0] = 0;

   hipMalloc((void **)&dev_keys, sizeof(Key)*nitems);
   hipMalloc((void **)&dev_vals, sizeof(Val)*nitems);

   hipMemcpy(dev_keys, host_keys, sizeof(Key)*nitems, hipMemcpyHostToDevice);
   hipMemcpy(dev_vals, host_vals, sizeof(Val)*nitems, hipMemcpyHostToDevice);

   Filter * test_filter = Filter::generate_on_device(Initializer, 42);

   printf("Test size: %llu\n", num_bits);

   hipDeviceSynchronize();

   find_first_fill<Filter, Key, Val><<<1, 32>>>(test_filter, dev_keys, dev_vals, nitems, returned_nitems);

   hipDeviceSynchronize();

   printf("Returned %llu\n", returned_nitems[0]);

   hipMemcpy(dev_keys, host_keys, sizeof(Key)*nitems, hipMemcpyHostToDevice);
   hipMemcpy(dev_vals, host_vals, sizeof(Val)*nitems, hipMemcpyHostToDevice);

   hipDeviceSynchronize();

   uint64_t new_nitems = returned_nitems[0];

   speed_query_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(new_nitems), test_filter->get_block_size(new_nitems)>>>(test_filter, dev_keys, dev_vals, new_nitems, &misses[0], &misses[1]);

   hipDeviceSynchronize();

   printf("Final misses: initial misses %llu %f wrong values %llu %f\n", misses[0], 1.0*misses[0]/new_nitems, misses[1], 1.0*misses[1]/new_nitems);

   hipDeviceSynchronize();

   hipFree(misses);

   hipFree(returned_nitems);

   Filter::free_on_device(test_filter);

   hipFree(dev_keys);
   hipFree(dev_vals);

}



// __host__ void test_first_fail(uint64_t num_bits){

//    //tcf_find_first_fill<tcf, uint16_t, uint16_t>(num_bits);

//    tcf_find_first_fill<tcf_no_back, uint64_t, uint16_t>(num_bits);

// }


__host__ void test_twelve_16(){

   printf("test_twelve_16\n");
   using tcf_twelve_16_1 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 1, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_16_2 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 2, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_16_4 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_16_8 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 8, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_16_16 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 16, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


   test_tcf_speed<tcf_twelve_16_1, uint16_t>("tcf_twelve_16_1 ", 28, 20);
   test_tcf_speed<tcf_twelve_16_2, uint16_t>("tcf_twelve_16_2 ", 28, 20);
   test_tcf_speed<tcf_twelve_16_4, uint16_t>("tcf_twelve_16_4 ", 28, 20);
   test_tcf_speed<tcf_twelve_16_8, uint16_t>("tcf_twelve_16_8 ", 28, 20);
   test_tcf_speed<tcf_twelve_16_16, uint16_t>("tcf_twelve_16_16", 28, 20);

   printf("\n\n\n");

}

__host__ void test_twelve_8(){

   printf("test_twelve_8\n");
   using tcf_twelve_8_1 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 1, 8, poggers::insert_schemes::blocked_bucket_insert<10>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_8_2 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 2, 8, poggers::insert_schemes::blocked_bucket_insert<10>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_8_4 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 8, poggers::insert_schemes::blocked_bucket_insert<10>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_8_8 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 8, 8, poggers::insert_schemes::blocked_bucket_insert<10>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   test_tcf_speed<tcf_twelve_8_1, uint16_t>("tcf_twelve_8_1 ", 28, 20);
   test_tcf_speed<tcf_twelve_8_2, uint16_t>("tcf_twelve_8_2 ", 28, 20);
   test_tcf_speed<tcf_twelve_8_4, uint16_t>("tcf_twelve_8_4 ", 28, 20);
   test_tcf_speed<tcf_twelve_8_8, uint16_t>("tcf_twelve_8_8 ", 28, 20);
   
   printf("\n\n\n");

}

__host__ void test_twelve_12(){

   printf("test_twelve_12\n");
   using tcf_twelve_12_1 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 1, 12, poggers::insert_schemes::blocked_bucket_insert<6>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_12_2 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 2, 12, poggers::insert_schemes::blocked_bucket_insert<6>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_12_4 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 12, poggers::insert_schemes::blocked_bucket_insert<6>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_12_8 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 8, 8, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_16_16 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 16, 16, poggers::insert_schemes::blocked_bucket_insert<5>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


   test_tcf_speed<tcf_twelve_12_1, uint16_t>("tcf_twelve_12_1 ", 28, 20);
   test_tcf_speed<tcf_twelve_12_2, uint16_t>("tcf_twelve_12_2 ", 28, 20);
   test_tcf_speed<tcf_twelve_12_4, uint16_t>("tcf_twelve_12_4 ", 28, 20);
   test_tcf_speed<tcf_twelve_12_8, uint16_t>("tcf_twelve_12_8 ", 28, 20);
   test_tcf_speed<tcf_twelve_16_16, uint16_t>("tcf_twelve_16_16", 28, 20);

   printf("\n\n\n");

}

//using tcf_twelve_32 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

__host__ void test_twelve_32(){

   printf("test_twelve_32\n");
   using tcf_twelve_32_1 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 1, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_32_2 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 2, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_32_4 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 4, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_32_8 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 8, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_32_16 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 16, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_twelve_32_32 = poggers::tables::bucketed_table<uint64_t,uint16_t, poggers::representations::wrapper_half_bucket<uint16_t>::representation, 32, 32, poggers::insert_schemes::blocked_bucket_insert<2>::representation, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


   test_tcf_speed<tcf_twelve_32_1, uint16_t>("tcf_twelve_32_1 ", 28, 20);
   test_tcf_speed<tcf_twelve_32_2, uint16_t>("tcf_twelve_32_2 ", 28, 20);
   test_tcf_speed<tcf_twelve_32_4, uint16_t>("tcf_twelve_32_4 ", 28, 20);
   test_tcf_speed<tcf_twelve_32_8, uint16_t>("tcf_twelve_32_8 ", 28, 20);
   test_tcf_speed<tcf_twelve_32_16, uint16_t>("tcf_twelve_32_16", 28, 20);
   test_tcf_speed<tcf_twelve_32_32, uint16_t>("tcf_twelve_32_32", 28, 20);

   printf("\n\n\n");

}

__host__ void test_sixteen_16(){

   printf("test_sixteen_16\n");
   using tcf_sixteen_16_1 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 1, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_16_2 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 2, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   using tcf_sixteen_16_4 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 4, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   using tcf_sixteen_16_8 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 8, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   using tcf_sixteen_16_16 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 16, 16, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;

   test_tcf_speed<tcf_sixteen_16_1, uint16_t>("tcf_sixteen_16_1 ", 28, 20);
   test_tcf_speed<tcf_sixteen_16_2, uint16_t>("tcf_sixteen_16_2 ", 28, 20);
   test_tcf_speed<tcf_sixteen_16_4, uint16_t>("tcf_sixteen_16_4 ", 28, 20);
   test_tcf_speed<tcf_sixteen_16_8, uint16_t>("tcf_sixteen_16_8 ", 28, 20);
   test_tcf_speed<tcf_sixteen_16_16, uint16_t>("tcf_sixteen_16_16", 28, 20);
   
   printf("\n\n\n");

}

__host__ void test_sixteen_32(){

   printf("test_sixteen_32\n");
   using tcf_sixteen_32_1 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 1, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_32_2 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 2, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_32_4 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 4, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_32_8 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 8, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_32_16 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 16, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_sixteen_32_32 = poggers::tables::static_table<uint64_t,uint16_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint16_t>::representation, 32, 32, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;


   test_tcf_speed<tcf_sixteen_32_1, uint16_t>("tcf_sixteen_32_1 ", 28, 20);
   test_tcf_speed<tcf_sixteen_32_2, uint16_t>("tcf_sixteen_32_2 ", 28, 20);
   test_tcf_speed<tcf_sixteen_32_4, uint16_t>("tcf_sixteen_32_4 ", 28, 20);
   test_tcf_speed<tcf_sixteen_32_8, uint16_t>("tcf_sixteen_32_8 ", 28, 20);
   test_tcf_speed<tcf_sixteen_32_16, uint16_t>("tcf_sixteen_32_16", 28, 20);
   test_tcf_speed<tcf_sixteen_32_32, uint16_t>("tcf_sixteen_32_32", 28, 20);

   printf("\n\n\n");

}


__host__ void test_eight_8(){

   printf("test_8_8\n");
   using tcf_8_8_1 = poggers::tables::static_table<uint64_t,uint8_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint8_t>::representation, 1, 8, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_8_8_2 = poggers::tables::static_table<uint64_t,uint8_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint8_t>::representation, 2, 8, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_8_8_4 = poggers::tables::static_table<uint64_t,uint8_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint8_t>::representation, 4, 8, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;
   using tcf_8_8_8 = poggers::tables::static_table<uint64_t,uint8_t, poggers::representations::dynamic_container<poggers::representations::key_container,uint8_t>::representation, 8, 8, poggers::insert_schemes::power_of_n_insert_shortcut_scheme, 2, poggers::probing_schemes::doubleHasher, poggers::hashers::murmurHasher>;




   test_tcf_speed<tcf_8_8_1, uint8_t>("tcf_8_8_1 ", 28, 20);
   test_tcf_speed<tcf_8_8_2, uint8_t>("tcf_8_8_2 ", 28, 20);
   test_tcf_speed<tcf_8_8_4, uint8_t>("tcf_8_8_4 ", 28, 20);
   test_tcf_speed<tcf_8_8_8, uint8_t>("tcf_8_8_8 ", 28, 20);

   printf("\n\n\n");

}






int main(int argc, char** argv) {

   // poggers::sizing::size_in_num_slots<1> first_size_20(1ULL << 20);
   // printf("2^20\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_20);

   // poggers::sizing::size_in_num_slots<1> first_size_22(1ULL << 22);
   // printf("2^22\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_22);

   // poggers::sizing::size_in_num_slots<1> first_size_24(1ULL << 24);
   // printf("2^24\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_24);

   // poggers::sizing::size_in_num_slots<1> first_size_26(1ULL << 26);
   // printf("2^26\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_26);

   // poggers::sizing::size_in_num_slots<1> first_size_28(1ULL << 28);
   // printf("2^28\n");
   // test_speed<table_type, uint64_t, uint64_t>(&first_size_28);


   // printf("alt table\n");

   // poggers::sizing::size_in_num_slots<1>half_split_20(6000);
   // test_speed<p2_table, key_type, val_type>(&half_split_20);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_22);

   // poggers::sizing::size_in_num_slots<2>half_split_24(1ULL << 23, 1ULL << 23);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_24);

   // poggers::sizing::size_in_num_slots<2>half_split_26(1ULL << 25, 1ULL << 25);
   // test_speed<small_double_type, uint64_t, uint64_t>(&half_split_26);


//   printf("P2 tiny table\n");
   // poggers::sizing::size_in_num_slots<1>half_split_28(1ULL << 28);
   // test_speed<p2_table, key_type, val_type>(&half_split_28);

   // poggers::sizing::variadic_size size(100000,100);
   // tcf * test_tcf = tcf::generate_on_device(&size, 42);


   // hipDeviceSynchronize();

   // tcf::free_on_device(test_tcf);


   // warpcore_bloom my_filter((1ULL << 20), 7);


   // test_bloom_speed("bloom_results/test", 20, 20, true);
   // test_bloom_speed("bloom_results/test", 22, 20, false);
   // test_bloom_speed("bloom_results/test", 24, 20, false);
   // test_bloom_speed("bloom_results/test", 26, 20, false);
   // test_bloom_speed("bloom_results/test", 28, 20, false);
   //test_bloom_speed("bloom_results/test", 30, 20, false);

   //test_tcf_speed("results/test", 10, 20, true);
   

   //test_tcf_speed("results/test", 6, 1, true);
   //test_tcf_speed("results/test", 20, 1, true);

   printf("Starting CG Tests\n");

   test_eight_8();

   test_twelve_8();

   test_twelve_12();

   test_twelve_16();

   test_twelve_32();

   test_sixteen_16();

   test_sixteen_32();

   

   hipDeviceSynchronize();

   printf("Tests over\n");



   // test_first_fail(22);
   // test_first_fail(24);
   // test_first_fail(26);
   // test_first_fail(28);
   // test_first_fail(30);


   return 0;

}
