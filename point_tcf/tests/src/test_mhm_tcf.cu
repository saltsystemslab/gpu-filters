#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 *
 *        About:
 *          This file contains speed tests for several Hash Table Types
 *          built using POGGERS. For more verbose testing please see the 
 *          benchmarks folder.
 *
 * ============================================================================
 */


#include <poggers/data_structs/tcf.cuh>

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <openssl/rand.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


// using backing_table = poggers::tables::bucketed_table<
//     uint64_t, uint8_t,
//     poggers::representations::dynamic_bucket_container<poggers::representations::dynamic_container<
//         poggers::representations::bit_grouped_container<10, 6>::representation, uint16_t>::representation>::representation,
//     1, 8, poggers::insert_schemes::linear_insert_bucket_scheme, 20, poggers::probing_schemes::doubleHasher,
//     poggers::hashers::murmurHasher>;
// using TCF = poggers::tables::bucketed_table<
//     uint64_t, uint8_t,
//     poggers::representations::dynamic_bucket_container<poggers::representations::dynamic_container<
//         poggers::representations::bit_grouped_container<10, 6>::representation, uint16_t>::representation>::representation,
//     1, 8, poggers::insert_schemes::power_of_n_insert_shortcut_bucket_scheme, 2, poggers::probing_schemes::doubleHasher,
//     poggers::hashers::murmurHasher, true, backing_table>;


using TCF = poggers::data_structs::tcf_wrapper<uint64_t, uint8_t, 26, 6, 1, 16>::tcf;


#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


uint64_t num_slots_per_p2(uint64_t nitems){

   //uint64_t nitems = .9*(1ULL << nbits);

   //for p=1/100, this is the correct value

   uint64_t nslots = 959*nitems/100;
   printf("using %llu slots\n", nslots);
   return nslots; 

}


template <typename T>
__host__ T * load_main_data(uint64_t nitems){


   char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/main_data-32-data.txt";

   //char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/main_data-32-data.txt";

   char * vals = (char * ) malloc(nitems * sizeof(T));

   //std::ifstream myfile(main_location);

   //std::string line;


   FILE * pFile;


   pFile = fopen(main_location, "rb");

   if (pFile == NULL) abort();

   size_t result;

   result = fread(vals, 1, nitems*sizeof(T), pFile);

   if (result != nitems*sizeof(T)) abort();



   // //current supported format is no spacing one endl for the file terminator.
   // if (myfile.is_open()){


   //    getline(myfile, line);

   //    strncpy(vals, line.c_str(), sizeof(uint64_t)*nitems);

   //    myfile.close();
      

   // } else {

   //    abort();
   // }


   return (T *) vals;


}

template <typename T>
__host__ T * load_alt_data(uint64_t nitems){


   char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/fp_data-32-data.txt";

   //char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/fp_data-32-data.txt";


   char * vals = (char * ) malloc(nitems * sizeof(T));


   //std::ifstream myfile(main_location);

   //std::string line;


   FILE * pFile;


   pFile = fopen(main_location, "rb");

   if (pFile == NULL) abort();

   size_t result;

   result = fread(vals, 1, nitems*sizeof(T), pFile);

   if (result != nitems*sizeof(T)) abort();



   return (T *) vals;


}

template <typename T>
__host__ T * generate_data(uint64_t nitems){


   //malloc space

   T * vals = (T *) malloc(nitems * sizeof(T));


   //          100,000,000
   uint64_t cap = 100000000ULL;

   for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

      uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


      RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



      to_fill += togen;

      //printf("Generated %llu/%llu\n", to_fill, nitems);

   }

   return vals;
}


template <typename Filter, typename Key, typename Val>
__global__ void find_first_fill(Filter * filter, Key * keys, Val * vals, uint64_t nitems, uint64_t * returned_nitems){


   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid != 0) return;

   // if (tile.thread_rank() == 0){

   //    for (int i = 0; i < 10; i++){
   //       printf("%d: %llu, %llu\n", i, keys[i], vals[i]);
   //    }
   // }


   //printf("Starting!\n");

   for (uint64_t i = 0; i < nitems; i++){


      if (!filter->insert(tile, keys[i])){

         if (tile.thread_rank() == 0){

            printf("Inserted %llu / %llu, %f full\n", i, nitems, 1.0*i/nitems);

         }

         returned_nitems[0] = i;

         return;

      } else {

         Val alt_val = 0;
         assert(filter->query(tile, keys[i], alt_val));
         assert(alt_val == vals[i]);


      }

      
   }

   if (tile.thread_rank() == 0) printf("All %llu items inserted\n", nitems);

}



template <typename Filter, typename Key, typename Val>
__global__ void speed_insert_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;


   if (!filter->insert(tile, keys[tid], vals[tid])){

      if (tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) misses, 1ULL);


   } else{

      Val test_val = 0;
      test_val +=0;
      assert(filter->query(tile, keys[tid], test_val));

      //assert(test_val == vals[tid]);
   }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

template <typename Filter, typename Key, typename Val>
__global__ void debug_insert_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * misses, bool * missed){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;


   if (!filter->insert(tile, keys[tid], vals[tid])){

      //filter->insert(tile, keys[tid], vals[tid]);

      if (tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) misses, 1ULL);

      missed[tid] = true;


   } else{

      Val test_val = 0;
      assert(filter->query(tile, keys[tid], test_val));

      missed[tid] = false;

      //assert(test_val == vals[tid]);
   }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

template <typename Filter, typename Key, typename Val>
__global__ void debug_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures, bool * missed){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   if (missed[tid]) return;

   Val test_val = 0;
   test_val +=0;

   if (!filter->query(tile,keys[tid], test_val)){


      filter->query(tile,keys[tid], test_val);


      if(tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) query_misses, 1ULL);

   } else {


      // if (test_val != vals[tid] && tile.thread_rank() == 0){
      //    atomicAdd((unsigned long long int *) query_failures, 1ULL);
      // }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Key, typename Val>
__global__ void speed_remove_kernel(Filter * filter, Key * keys, uint64_t nvals, uint64_t * misses){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;




   if (!filter->remove(tile, keys[tid]) && tile.thread_rank() == 0){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   } 
      //else{

   //    Val test_val = 0;
   //    assert(filter->query(tile, keys[tid], test_val));
   // }

   //assert(filter->insert(tile, keys[tid], vals[tid]));


}

__global__ void count_bf_misses(bool * vals, uint64_t nitems, uint64_t * misses){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= nitems) return;


   if (!vals[tid]){
      atomicAdd((unsigned long long int *) misses, 1ULL);
   }
}

template <typename Filter, typename Key, typename Val>
__global__ void speed_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   Val test_val = 0;

   if (!filter->query(tile,keys[tid], test_val)){


      //filter->query(tile,keys[tid], test_val);


      if(tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) query_misses, 1ULL);

   } else {


      if (test_val != vals[tid] && tile.thread_rank() == 0){
         atomicAdd((unsigned long long int *) query_failures, 1ULL);
      }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Key, typename Val>
__global__ void fp_speed_query_kernel(Filter * filter, Key * keys, Val * vals, uint64_t nvals, uint64_t * query_misses, uint64_t * query_failures){

   auto tile = filter->get_my_tile();

   uint64_t tid = tile.meta_group_size()*blockIdx.x + tile.meta_group_rank();

   if (tid >= nvals) return;

   Val test_val = 0;

   if (!filter->query(tile,keys[tid], test_val)){


   //    filter->query(tile,keys[tid], test_val);


      if(tile.thread_rank() == 0)
      atomicAdd((unsigned long long int *) query_misses, 1ULL);

   // } else {


      // if (test_val != vals[tid] && tile.thread_rank() == 0){
      //    atomicAdd((unsigned long long int *) query_failures, 1ULL);
      // }

   }
   //assert(filter->query(tile, keys[tid], val));


}


template <typename Filter, typename Val>
__host__ void test_tcf_speed(const std::string& filename, int num_bits, int num_batches){


   using Key = uint64_t;
   //using Val = uint8_t;

   //using Filter = tcf;

   //std::cout << "Starting " << filename << " " << num_bits << std::endl;

   // poggers::sizing::size_in_num_slots<2> pre_init ((1ULL << num_bits), (1ULL << num_bits)/100);

   // poggers::sizing::size_in_num_slots<2> * Initializer = &pre_init;

   uint64_t table_nitems = (1ULL << num_bits);

   poggers::sizing::size_in_num_slots<2> pre_init((table_nitems * 90ULL / 100ULL), (table_nitems * 10ULL / 100ULL));
   //poggers::sizing::size_in_num_slots<2> pre_init (table_nitems*.9, table_nitems*.1);

   poggers::sizing::size_in_num_slots<2> * Initializer = &pre_init;



   uint64_t nitems = Initializer->total()*.9;

   Key * host_keys = generate_data<Key>(nitems);
   Val * host_vals = generate_data<Val>(nitems);


   //For MHM TCF - we must clip keys to range [0,2^6);
   //otherwise the fp rate looks super high

   for (uint64_t i = 0; i < nitems; i++){
      host_vals[i] = host_vals[i] % 64;
   }


   Key * fp_keys = generate_data<Key>(nitems);

   Key * dev_keys;

   Val * dev_vals;




   uint64_t * misses;

   hipMallocManaged((void **)& misses, sizeof(uint64_t)*5);
   hipDeviceSynchronize();

   //printf("Data generated\n");

   misses[0] = 0;
   misses[1] = 0;
   misses[2] = 0;
   misses[3] = 0;
   misses[4] = 0;

   //static seed for testing
   Filter * test_filter = Filter::generate_on_device(Initializer, 42);

   hipDeviceSynchronize();

   //init timing materials
   std::chrono::duration<double>  * insert_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
   std::chrono::duration<double>  * query_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
   std::chrono::duration<double>  * fp_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));

   std::chrono::duration<double>  * delete_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));



   uint64_t * batch_amount = (uint64_t *) malloc(num_batches*sizeof(uint64_t));

   //print_tid_kernel<Filter, Key, Val><<<test_filter->get_num_blocks(nitems),test_filter->get_block_size(nitems)>>>(test_filter, dev_keys, dev_vals, nitems);


   for (uint64_t i = 0; i < num_batches; i++){

      uint64_t start_of_batch = i*nitems/num_batches;
      uint64_t items_in_this_batch = (i+1)*nitems/num_batches;

      if (items_in_this_batch > nitems) items_in_this_batch = nitems;

      items_in_this_batch = items_in_this_batch - start_of_batch;


      batch_amount[i] = items_in_this_batch;


      hipMalloc((void **)& dev_keys, items_in_this_batch*sizeof(Key));
      hipMalloc((void **)& dev_vals, items_in_this_batch*sizeof(Val));


      hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      bool * missed;

      hipMalloc((void **)&missed, items_in_this_batch*sizeof(bool));


      uint64_t num_blocks = test_filter->get_num_blocks(items_in_this_batch);
      uint64_t block_size = test_filter->get_block_size(items_in_this_batch);



      //ensure GPU is caught up for next task
      hipDeviceSynchronize();

      auto insert_start = std::chrono::high_resolution_clock::now();

      //add function for configure parameters - should be called by ht and return dim3
      hipLaunchKernelGGL(HIP_KERNEL_NAME(speed_insert_kernel<Filter, Key, Val>), num_blocks, block_size, 0, 0, test_filter, dev_keys, dev_vals, items_in_this_batch, misses);
   

      hipDeviceSynchronize();
      auto insert_end = std::chrono::high_resolution_clock::now();

      insert_diff[i] = insert_end-insert_start;

      hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      hipDeviceSynchronize();

      auto query_start = std::chrono::high_resolution_clock::now();

      hipLaunchKernelGGL(HIP_KERNEL_NAME(speed_query_kernel<Filter, Key, Val>), num_blocks, block_size, 0, 0, test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[1], &misses[2]);
      
      
      hipDeviceSynchronize();
      auto query_end = std::chrono::high_resolution_clock::now();


     
      query_diff[i] = query_end - query_start;

      hipMemcpy(dev_keys, fp_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
      hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);


      hipDeviceSynchronize();

      auto fp_start = std::chrono::high_resolution_clock::now();

      hipLaunchKernelGGL(HIP_KERNEL_NAME(speed_query_kernel<Filter, Key, Val>), num_blocks, block_size, 0, 0, test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[3], &misses[4]);


      hipDeviceSynchronize();
      auto fp_end = std::chrono::high_resolution_clock::now();

      fp_diff[i] = fp_end-fp_start;


      hipFree(dev_keys);
      hipFree(dev_vals);

      hipFree(missed);


   }

   //deletes
   // for (uint64_t i = 0; i < num_batches; i++){

   //    uint64_t start_of_batch = i*nitems/num_batches;
   //    uint64_t items_in_this_batch = (i+1)*nitems/num_batches;

   //    if (items_in_this_batch > nitems) items_in_this_batch = nitems;

   //    items_in_this_batch = items_in_this_batch - start_of_batch;


   //   // batch_amount[i] = items_in_this_batch;


   //    hipMalloc((void **)& dev_keys, items_in_this_batch*sizeof(Key));
   //    //hipMalloc((void **)& dev_vals, items_in_this_batch*sizeof(Val));


   //    hipMemcpy(dev_keys, host_keys+start_of_batch, items_in_this_batch*sizeof(Key), hipMemcpyHostToDevice);
   //    //hipMemcpy(dev_vals, host_vals+start_of_batch, items_in_this_batch*sizeof(Val), hipMemcpyHostToDevice);

   //    hipDeviceSynchronize();

   //    auto delete_start = std::chrono::high_resolution_clock::now();

   //    hipLaunchKernelGGL(HIP_KERNEL_NAME(speed_query_kernel<Filter, Key, Val>), num_blocks, block_size, 0, 0, test_filter, dev_keys, dev_vals, items_in_this_batch, &misses[1], &misses[2]);
   //    hipDeviceSynchronize();
   //    auto delete_end = std::chrono::high_resolution_clock::now();


     
   //    delete_diff[i] = delete_end - delete_start;

   // }

   hipDeviceSynchronize();


   Filter::free_on_device(test_filter);

   free(host_keys);
   free(host_vals);
   free(fp_keys);

   //free pieces

   //time to output


   printf("nitems: %llu, insert misses: %llu, query missed: %llu, query mismatch %llu, fp missed %llu, fp mismatch %llu\n", nitems, misses[0], misses[1], misses[2], misses[3], misses[4]);

   std::chrono::duration<double> summed_insert_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_insert_diff += insert_diff[i];
   }

   std::chrono::duration<double> summed_query_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_query_diff += query_diff[i];
   }

   std::chrono::duration<double> summed_fp_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_fp_diff += fp_diff[i];
   }

   std::chrono::duration<double> summed_delete_diff = std::chrono::nanoseconds::zero();

   for (int i =0; i < num_batches;i++){
      summed_delete_diff += delete_diff[i];
   }

  


   const uint64_t scaling_factor = 1000000ULL;



   double insert_throughput = nitems/(scaling_factor*summed_insert_diff.count());
      
   double lookup_throughput = nitems/(scaling_factor*summed_query_diff.count());

   double fp_throughput = nitems/(scaling_factor*summed_fp_diff.count());

   std::cout << "Name,  insert perf (M/s), lookup perf (M/s), FP perf (M/s)" << std::endl;
   std::cout << filename << ": " << insert_throughput << ", " << lookup_throughput << ", " << fp_throughput <<"." << std::endl;

   // std::cout << insert_file << std::endl;
   return;




}



int main(int argc, char** argv) {

   printf("Starting tests\n");

   test_tcf_speed<TCF, uint8_t>("tcf_mhm_20", 20, 20);

   test_tcf_speed<TCF, uint8_t>("tcf_mhm_22", 22, 20);

   test_tcf_speed<TCF, uint8_t>("tcf_mhm_24", 24, 20);

   test_tcf_speed<TCF, uint8_t>("tcf_mhm_26", 26, 20);

   test_tcf_speed<TCF, uint8_t>("tcf_mhm_28", 28, 20);
   

   hipDeviceSynchronize();

   printf("Tests over\n");



   // test_first_fail(22);
   // test_first_fail(24);
   // test_first_fail(26);
   // test_first_fail(28);
   // test_first_fail(30);


   return 0;

}
